
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
//#include<cuda.h>
#include<unistd.h>
#include<time.h>


__global__ void multiply(int *scval, int *sccol, int *vec, int *result, int *cols, int *rowptr)
{
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
        int sum=0;
        int i;
        int colidx=tid/2;

   	for(i=0;i<cols[colidx];i++)
	{	

		sum += vec[sccol[rowptr[tid]+i]]*scval[rowptr[tid]+i];

	}
     	__syncthreads();
   	result[tid]=sum;
}

__global__ void printmatscreen(int* mat, int N)
{
	int i;
	for (i=0;i<N;i++)
	{	
		printf("%d ",mat[i]);
		
	}
	printf("\n");
}



int** Make2DIntArray(int arraySizeX, int arraySizeY)
{
	int** theArray;
	theArray = (int**) malloc(arraySizeX*sizeof(int*));
	int i;
	for (i = 0; i < arraySizeX; i++)
	   theArray[i] = (int*) malloc(arraySizeY*sizeof(int));
	int j;
	for (i=0;i<arraySizeX;i++)
	{
	    for (j=0;j<arraySizeY;j++)
	    {
	        theArray[i][j]=0;
	    }
	}
	return theArray;
}

int** Make2DVariableIntArray(int rows, int blocks, int blocksize, int* columns)
{
int** theArray;
theArray = (int**) malloc(rows*sizeof(int*));
int i, j, k;
for (i = 0; i < blocks; i++)
{
	k=columns[i];
	for (j=0; j < blocksize; j++)
	{
		theArray[i*blocksize+j] = (int*) malloc(k*sizeof(int));
	}
   
}
//int j;

for (i=0;i<blocks;i++)
{
    for (j=0;j<blocksize;j++)
    {
    	for (k=0;k<columns[i];k++)
	{
	        theArray[i*blocksize+j][k]=0;
    	}
    }
}

   return theArray;
}

int** Changeto2DVariableIntArray(int** theArray,int rows, int blocks, int blocksize, int* columns)
{

int** NewArray=Make2DVariableIntArray(rows,blocks,blocksize,columns);
int i, j, k;

for (i=0;i<blocks;i++)
{
    for (j=0;j<blocksize;j++)
    {
        for (k=0;k<columns[i];k++)
        {
                NewArray[i*blocksize+j][k]=theArray[i*blocksize+j][k];
        }
    }
}
printf("changed to multiple matrixes");
   return NewArray;
}

void init_zeros(int** matrix, int N)
{
	int i,j;
	for (i=0;i<N;i++)
	{	
		for (j=0;j<N;j++)
		{
			matrix[i][j]=0;
		}
	}
}

void printmat(int** matrix, int N, int Nj)
{
	int i,j;
	
	for (i=0;i<N;i++)
	{	
		printf("\n");
		for (j=0;j<N;j++)
		{
			printf("%d ",matrix[i][j]);
		}
	}
	printf("\n");
}

void printtofile(int** matrix, int K, char* filename)
{
	/*
	Prints original 2D matrices to file
	*/
	FILE *fp;
	fp=fopen(filename,"wt");
	int i,j;
	for (i=0;i<K;i++)
	{
		fprintf(fp, "\n");
		for (j=0;j<K;j++)
		{
			fprintf(fp, "%d\t", matrix[i][j]);
		}
	}
}

void printtofile1D(int* matrix, int K, char* filename)
{
	/*
	Prints resultant matrix to a file
	*/
	FILE *fp;
	fp=fopen(filename,"wt");
	int i;
	
	for (i=0;i<K;i++)
	{
		fprintf(fp, "%d\n", matrix[i]);
		
	}
	
}

int* Make1DIntArray(int arraySizeX)
{
int* theArray;
theArray = (int*)malloc(arraySizeX*sizeof(int));
int i;
for (i=0;i<arraySizeX;i++)
{
    theArray[i]=0;
}

   return theArray;
}

void freese(int sizeX, int sizeY, double** ptr)
{
    int i;
     for (i=0;i<sizeX;i++)
        free(ptr[i]);
    free(ptr);
}

int main()
{
	int N=1000;
	
//	const int Dsize=1000;
	FILE *arr, *vec;
	int i,j,maxrowwidth=0,tint=0;
	int** a=Make2DIntArray(N,N);
//	int* val=Make1DIntArray(Dsize);
//	int* col=Make1DIntArray(Dsize);
//	int* row=Make1DIntArray(Dsize);
	int* result=Make1DIntArray(N);
	int* vecX=Make1DIntArray(N);
	int** scval=Make2DIntArray(N,N);    //sell c value
	int** sccol=Make2DIntArray(N,N);	//sell c col
	int* rowwidth=Make1DIntArray(N);	//number of elements in each row
	int* temp=Make1DIntArray(N);
	int* rows=Make1DIntArray(N);
	int* resultsordered=Make1DIntArray(N);
	
	int sig=4,c=2;
//	int* rowwidth=Make1DIntArray(N);
	int *dev_vec, *dev_scval, *dev_result, *dev_sccol, *dev_cols, *dev_rowptr;
	
	//int val[10],col[10],row[10];
	arr=fopen("matrix100.txt","r");
	int k=0;
//	struct timeval start, end;
//	gettimeofday(&start, NULL);
	
	
	//Reading the vector
	
	vec=fopen("vector100.txt","r");
	for (i=0;i<N;i++)
	{
		fscanf(vec,"%d",&vecX[i]);
		rows[i]=i;
	}
	
	//Reading the matrix
	
	for(i=0;i<N;i++)
	{	
		printf("\n");
		for(j=0;j<N;j++)
		{
			fscanf(arr,"%d",&a[i][j]);
//			printf("%d ",a[i][j]);
		}
	}
	printf("\n");
	//row[i]=k;
       	//printf("\n k = %d\n ", k);
       	//sleep(10);
//	gettimeofday(&end, NULL);

//	double delta = ((end.tv_sec  - start.tv_sec) * 1000000u + 
//	         end.tv_usec - start.tv_usec) / 1.e6;

//	printf("\nTime spent=%f\n", delta);	

	
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{	
			if(a[i][j]!=0)
			{
				scval[i][k]=a[i][j];

				sccol[i][k]=j;
				rowwidth[i]=k+1;
				if(rowwidth[i]>maxrowwidth)
				{
					maxrowwidth=rowwidth[i];
				}k++;
			}
				
		}
		//printf("\nRow width %d = %d", i, rowwidth[i]);
		k=0;
	}
	

if(sig>1&&c!=sig)	
{	
	for(i=0;i<N;i=i+sig)
	{
		for(k=0;k<sig-1;k++)
		{
		for(j=i;(j<i+sig-1) && (j<N);j++)
		{
			if(rowwidth[j]<rowwidth[j+1])
			{	
				temp=scval[j];	
				scval[j]=scval[j+1];
				scval[j+1]=temp;
				temp=sccol[j];
				sccol[j]=sccol[j+1];
				sccol[j+1]=temp;
				tint=rowwidth[j];
				rowwidth[j]=rowwidth[j+1];
				rowwidth[j+1]=tint;
				tint=rows[j];
				rows[j]=rows[j+1];
				rows[j+1]=tint;
			}
		}
		}
		}	
}	


/*	for(i=0;i<N;i++)
	{
		if(scval[i][0]==0)
		{
			break;	
		}	
	}
	N=i;
*/		
	printf("\nmaxrowwidth=%d\n",maxrowwidth);
//	printmat(scval,N,N);
//	printtofile(scval,N,"scval.txt");
//	printtofile(sccol,N,"sccol.txt");
/*	printf("\n Shuffled rows is:\n");
	for (i=0;i<N;i++)
	{
		printf("%d\n",rows[i]);
	}
	
*/
        
        //printmatscreen<<<1,1>>>(dev_b,N);
/*	multiply<<<N,N>>>(dev_a, dev_b, dev_c, N, N);
	
	cudaMemcpy(result, dev_c, sizeof(int)*N, cudaMemcpyDeviceToHost);

        for (i=0;i<N;i++)
        {
        	printf("\n%d",result[i]);
        }
        
        cudaFree(dev_a);
        cudaFree(dev_b);
        cudaFree(dev_c);
	
	// NEED TO FIGURE OUT A WAY TO POPULATE cols SO AS TO HAVE varmat CREATED PROPERLY. SYSTEM CRASHES OTHERWISE
*/	
	int* cols=Make1DIntArray(N/c);
	j=0;
	int colsum=0;
	for(i=0;i<N;i=i+c)
	{
		cols[j]=rowwidth[i];
		colsum+=cols[j];
		j++;
		
	}
	
	int** varscval=Changeto2DVariableIntArray(scval,N,N/c,c,cols);
	int** varsccol=Changeto2DVariableIntArray(sccol,N,N/c,c,cols);
	for (i=0;i<N/c;i++)
	{
		for(j=0;j<c;j++)
		{
			printf("\n");
			for (k=0;k<cols[i];k++)
			{
				printf("%d ",varscval[i*c+j][k]);
				//printf("%d ",varsccol[i*c+j][k]);
				
			}
		}
	}

	int varsize=colsum*c;

	//flattening scval and sccol
	int counters=0;
	int* scval_flat=Make1DIntArray(varsize);
	int* sccol_flat=Make1DIntArray(varsize);
	int* rowptr=Make1DIntArray(N+1);
	rowptr[0]=0;	
	int countcols=0;
	int z=0;
	for (i=0;i<N/c;i++)
	{
		for(j=0;j<c;j++)
		{
			
			countcols=0;
			for (k=0;k<cols[i];k++)
			{
				
				scval_flat[counters]=varscval[i*c+j][k];
				sccol_flat[counters]=varsccol[i*c+j][k];
				counters=counters+1;
				countcols=countcols+1;
			}
			rowptr[z+1]=rowptr[z]+countcols;
			z=z+1;
		}
	}
	
	printtofile(scval,N,"scval.txt");
	printtofile(sccol,N,"sccol.txt");

	hipEvent_t start, stop, start_kernel, stop_kernel;
	float time, time_kernel;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);
	
	hipMalloc((void**)&dev_vec, sizeof(int)*N);
        hipMalloc((void**)&dev_scval, sizeof(int)*varsize);
        hipMalloc((void**)&dev_result, sizeof(int)*N);
        hipMalloc((void**)&dev_sccol, sizeof(int)*varsize);	
        hipMalloc((void**)&dev_cols, sizeof(int)*(N/c));
        hipMalloc((void**)&dev_rowptr, sizeof(int)*N);
	
	hipEventRecord(start,0);
		
	hipMemcpy(dev_vec, vecX, sizeof(int)*N, hipMemcpyHostToDevice);
        hipMemcpy(dev_scval, scval_flat, sizeof(int)*varsize, hipMemcpyHostToDevice);
        hipMemcpy(dev_result, result, sizeof(int)*N, hipMemcpyHostToDevice);
        hipMemcpy(dev_sccol, sccol_flat, sizeof(int)*varsize, hipMemcpyHostToDevice);
        hipMemcpy(dev_cols, cols, sizeof(int)*(N/c), hipMemcpyHostToDevice);
        hipMemcpy(dev_rowptr, rowptr, sizeof(int)*N, hipMemcpyHostToDevice);
	
	hipEventRecord(start_kernel,0);	

	multiply<<<N/c,c>>>(dev_scval, dev_sccol, dev_vec, dev_result, dev_cols, dev_rowptr);

	hipEventRecord(stop_kernel,0);

	hipMemcpy(result, dev_result, sizeof(int)*N, hipMemcpyDeviceToHost);
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);
	hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel);
	
	printf("\nTime for kernel with data transfer = %f ms \n", time);
	printf("\nTime for kernel without data transfer = %f ms \n", time_kernel); 
	
        for (i=0;i<N;i++)
        {
        	resultsordered[rows[i]]=result[i];
        }
        
        printtofile1D(resultsordered,N,"results.txt");
        
        // CODE TO RESHUFFLE BACK

	
	hipFree(dev_vec);
        hipFree(dev_scval);
        hipFree(dev_result);
	hipFree(dev_sccol);
	hipFree(dev_cols);
	return 0;



/*	
	cudaMalloc((void**)&dev_vec, sizeof(int)*N);
        cudaMalloc((void**)&dev_scval_flat, sizeof(int)*varsize);
        cudaMalloc((void**)&dev_result, sizeof(int)*N);
        cudaMalloc((void**)&dev_sccol_flat, sizeof(int)*varsize);	
		
	cudaMemcpy(dev_a, vecX, sizeof(int)*N, cudaMemcpyHostToDevice);
        cudaMemcpy(dev_b, varscval, sizeof(int)*varsize, cudaMemcpyHostToDevice);
        cudaMemcpy(dev_c, result, sizeof(int)*N, cudaMemcpyHostToDevice);
        cudaMemcpy(dev_d, varsccol, sizeof(int)*varsize, cudaMemcpyHostToDevice);
*/	
}


        
        


