#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<time.h>
#include<hip/hip_runtime.h>
/*
__global__ void multiply(int *val, int *vec, int *result, int *cols, int *rowptr)
{
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
        int sum=0;
        int i;
        for(i=0;i<cols[colidx];i++)
	{	

		sum += vec[rowptr[tid]+i]*val[rowptr[tid]+i];

	}
     	__syncthreads();
   	result[tid]=sum;
}

__global__ void printmatscreen(int* mat, int N)
{
	int i;
	for (i=0;i<N;i++)
	{	
		printf("%d ",mat[i]);
		
	}
	printf("\n");
}
*/
int** Make2DIntArray(int arraySizeX, int arraySizeY)
{
	int** theArray;
	theArray = (int**) malloc(arraySizeX*sizeof(int*));
	int i;
	for (i = 0; i < arraySizeX; i++)
  	 theArray[i] = (int*) malloc(arraySizeY*sizeof(int));
	int j;

	for (i=0;i<arraySizeX;i++)
	{
    	for (j=0;j<arraySizeY;j++)
    	{
       	 theArray[i][j]=0;
    	}
}

   return theArray;
}

void init_zeros(int** matrix, int N)
{
	int i,j;
	for (i=0;i<N;i++)
	{	
		for (j=0;j<N;j++)
		{
			matrix[i][j]=0;
		}
	}
}

void printmat(int** matrix, int N)
{
	int i,j;
	
	for (i=0;i<N;i++)
	{	
		printf("\n");
		for (j=0;j<N;j++)
		{
			printf("%d \t",matrix[i][j]);
		}
	}
	printf("\n");
}


int* Make1DIntArray(int arraySizeX) {
int* theArray;
theArray = (int*)malloc(arraySizeX*sizeof(int));
int i;
for (i=0;i<arraySizeX;i++)
{
    theArray[i]=0;
}

   return theArray;
}

void freese(int sizeX, int sizeY, double** ptr)
{
    int i;
     for (i=0;i<sizeX;i++)
        free(ptr[i]);
    free(ptr);
}

void printtofile1D(int* matrix, int K, char* filename)
{
	/*
	Prints resultant matrix to a file
	*/
	FILE *fp;
	fp=fopen(filename,"wt");
	int i;
	
	for (i=0;i<K;i++)
	{
		fprintf(fp, "%d\n", matrix[i]);
		
	}
	
}

int main()
{

	const int N=10000;

	FILE *arr, *vec;
	int i,j;
	int** a=Make2DIntArray(N,N);
	
	int* result=Make1DIntArray(N);
	int* vecX=Make1DIntArray(N);
	//int val[10],col[10],row[10];
	int* resultsordered=Make1DIntArray(N);
	
//	int sig=4,c=2;
//	int* rowwidth=Make1DIntArray(N);
//	int *dev_vec, *dev_scval, *dev_result, *dev_sccol, *dev_cols, *dev_rowptr;
	
	arr=fopen("matrix10000.txt","r");
	int k=0,cinrow=0;

	//row[0]=0;
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			fscanf(arr,"%d",&a[i][j]);
			if(a[i][j])
			{
				k++;
			}	
			
		}
		cinrow=0;
		
	}

	
	int Dsize=k;
	int* val=Make1DIntArray(Dsize);
	int* col=Make1DIntArray(Dsize);
	int* row=Make1DIntArray(N+1);
       	printf("\n k = %d\n ", k);
       	//sleep(10);
       	
k=0;
	row[0]=0;
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			if(a[i][j])
			{
				val[k]=a[i][j];
				col[k]=j;
				cinrow++;
				k++;
				
			}	
			
		}
		row[i+1]=row[i]+cinrow;
		cinrow=0;
		
	}
//	sleep(5);
	row[i]=k;

	vec=fopen("vector10000.txt","r");
	for (i=0;i<N;i++)
	{
		fscanf(vec,"%d",&vecX[i]);
	
	
			
		
	}
	

	
	
	/*printf("\n Vector is:\n");
	for (i=0;i<N;i++)
	{
		printf("%d\n",vecX[i]);
	}
//printing val, col and row
	/*
	printf("Val=");
	for(i=0;i<Dsize;i++)
	{
		printf("%d\t",val[i]);
	}
	printf("\n");
	
	printf("col=");
        for(i=0;i<Dsize;i++)
        {
                printf("%d\t",col[i]);
        }
        printf("\n");
	printf("row=");
	for(i=0;i<k;i++)
        {
                printf("%d\t",row[i]);
        }
        */
        printf("\n");


        /*Now the actual multiplication kernel*/
        
	/*struct timeval start, end;
	gettimeofday(&start, NULL);        
	*/
	
	hipEvent_t start_kernel, stop_kernel;
	float time_kernel;
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);
	
	hipEventRecord(start_kernel,0);
	
       	for (i=0;i<N;i++)
        {
        	for (j=row[i];j<row[i+1];j++)
        	{ 	
        		result[i]+=val[j]*vecX[col[j]];
        		
        	}
       	}

	hipEventRecord(stop_kernel,0);




/*	gettimeofday(&end, NULL);

	double delta = ((end.tv_sec  - start.tv_sec) * 1000000u + 
	         end.tv_usec - start.tv_usec) / 1.e6;

	printf("\nTime spent=%f\n", delta);	

*/	hipEventSynchronize(stop_kernel);

//	multiply<<<N,1>>>(val, vecX, result, col, row);

	hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel);
	
	printf("\nTime for kernel without data transfer = %f ms \n", time_kernel); 
	
	printtofile1D(result,N,"results.txt");

/*	printf("\n Result is:\n");
	for (i=0;i<N;i++)
	{
		printf("%d\n",result[i]);
	}      
*/
	return 0;

}

