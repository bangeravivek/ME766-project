#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<unistd.h>
#include<time.h>

__global__ void multiply(int *vec, int *mat, int *out, const int N, const int M)
{
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
        int sum=0;
        
	if(tid<M)
	{
        	for(int i=0; i<N; i++)
        	{	
        		sum += vec[i]*mat[(tid*M)+i];
        	}
        	
   	}
   	out[tid]=sum;
}

__global__ void printmatscreen(int* mat, int N)
{
	int i,j;
	for (i=0;i<N;i++)
	{	
		printf("\n");
		for (j=0;j<N;j++)
		{
			printf("%d ",mat[(i*N)+j]);
		}
	}
	printf("\n");
}


int** Make2DIntArray(int arraySizeX, int arraySizeY)
{
	int** theArray;
	theArray = (int**) malloc(arraySizeX*sizeof(int*));
	int i;
	for (i = 0; i < arraySizeX; i++)
	   theArray[i] = (int*) malloc(arraySizeY*sizeof(int));
	int j;
	for (i=0;i<arraySizeX;i++)
	{
	    for (j=0;j<arraySizeY;j++)
	    {
	        theArray[i][j]=0;
	    }
	}
	return theArray;
}

int** Make2DVariableIntArray(int rows, int blocks, int blocksize, int* columns)
{
int** theArray;
theArray = (int**) malloc(rows*sizeof(int*));
int i, j, k;
for (i = 0; i < blocks; i++)
{
	k=columns[i];
	for (j=0; j < blocksize; j++)
	{
		theArray[i*blocksize+j] = (int*) malloc(k*sizeof(int));
	}
   
}
//int j;

for (i=0;i<blocks;i++)
{
    for (j=0;j<blocksize;j++)
    {
    	for (k=0;k<columns[i];k++)
	{
	        theArray[i*blocksize+j][k]=0;
    	}
    }
}

   return theArray;
}

int** Changeto2DVariableIntArray(int** theArray,int rows, int blocks, int blocksize, int* columns)
{

int** NewArray=Make2DVariableIntArray(rows,blocks,blocksize,columns);
int i, j, k;

for (i=0;i<blocks;i++)
{
    for (j=0;j<blocksize;j++)
    {
        for (k=0;k<columns[i];k++)
        {
                NewArray[i*blocksize+j][k]=theArray[i*blocksize+j][k];
        }
    }
}
printf("changed to multiple matrixes");
   return NewArray;
}

void init_zeros(int** matrix, int N)
{
	int i,j;
	for (i=0;i<N;i++)
	{	
		for (j=0;j<N;j++)
		{
			matrix[i][j]=0;
		}
	}
}

void printmat(int** matrix, int N, int Nj)
{
	int i,j;
	
	for (i=0;i<N;i++)
	{	
		printf("\n");
		for (j=0;j<N;j++)
		{
			printf("%d ",matrix[i][j]);
		}
	}
	printf("\n");
}

void printtofile(int** matrix, int K, char* filename)
{
	/*
	Prints original 2D matrices to file
	*/
	FILE *fp;
	fp=fopen(filename,"wt");
	int i,j;
	for (i=0;i<K;i++)
	{
		fprintf(fp, "\n");
		for (j=0;j<K;j++)
		{
			fprintf(fp, "%d\t", matrix[i][j]);
		}
	}
}

void printtofile1D(int* matrix, int K, char* filename)
{
	/*
	Prints resultant matrix to a file
	*/
	FILE *fp;
	fp=fopen(filename,"wt");
	int i,j;
	int counters=0;
	for (i=0;i<K;i++)
	{
		fprintf(fp, "\n");
		for (j=0;j<K;j++)
		{
			fprintf(fp, "%d \t", matrix[counters]);
			counters++;
		}
	}
}

int* Make1DIntArray(int arraySizeX)
{
int* theArray;
theArray = (int*)malloc(arraySizeX*sizeof(int));
int i;
for (i=0;i<arraySizeX;i++)
{
    theArray[i]=0;
}

   return theArray;
}

void freese(int sizeX, int sizeY, double** ptr)
{
    int i;
     for (i=0;i<sizeX;i++)
        free(ptr[i]);
    free(ptr);
}

void main()
{
	int N=100;
	
//	const int Dsize=1000;
	FILE *arr, *vec;
	int i,j,maxrowwidth=0,tint=0;
	int** a=Make2DIntArray(N,N);
//	int* val=Make1DIntArray(Dsize);
//	int* col=Make1DIntArray(Dsize);
//	int* row=Make1DIntArray(Dsize);
	int* result=Make1DIntArray(N);
	int* vecX=Make1DIntArray(N);
	int** scval=Make2DIntArray(N,N);    //sell c value
	int** sccol=Make2DIntArray(N,N);	//sell c col
	int* rowwidth=Make1DIntArray(N);	//number of elements in each row
	int* temp=Make1DIntArray(N);
	int *dev_a, *dev_b, *dev_c, *dev_d;
	
	//int val[10],col[10],row[10];
	arr=fopen("matrix100.txt","r");
	int k=0,cinrow=0;
	struct timeval start, end;
	gettimeofday(&start, NULL);
	
	
	//Reading the vector
	
	vec=fopen("vector100.txt","r");
	for (i=0;i<N;i++)
	{
		fscanf(vec,"%d",&vecX[i]);
	}
	
	//Reading the matrix
	
	for(i=0;i<N;i++)
	{	
		printf("\n");
		for(j=0;j<N;j++)
		{
			fscanf(arr,"%d",&a[i][j]);
			printf("%d ",a[i][j]);
		}
	}
	printf("\n");
	//row[i]=k;
       	//printf("\n k = %d\n ", k);
       	//sleep(10);
	gettimeofday(&end, NULL);

	double delta = ((end.tv_sec  - start.tv_sec) * 1000000u + 
	         end.tv_usec - start.tv_usec) / 1.e6;

	printf("\nTime spent=%f\n", delta);	

	
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{	
			if(a[i][j]!=0)
			{
				scval[i][k]=a[i][j];
				//printf("\n scval[%d][%d]=%d",i,k,scval[i][k]);

				//sleep(1);
				sccol[i][k]=j;
				//printf("\n sccol[%d][%d]=%d",i,k,sccol[i][k]);
				rowwidth[i]=k+1;
				if(rowwidth[i]>maxrowwidth)
				{	//printf("\nrow[%d] width=%d\n",i,maxrowwidth);
					maxrowwidth=rowwidth[i];
				}k++;
			}
				
		}
		//printf("\nRow width %d = %d", i, rowwidth[i]);
		k=0;
	}
	
	
	for(i=0;i<N-1;i++)
	{
	for(j=0;j<N-1;j++)
	{
		if(rowwidth[j]<rowwidth[j+1])
		{	/*printf("\nrow %d width=%d",j,rowwidth[j]);
			printf("\nscval[%d]=",j);
			for(k=0;k<rowwidth[j];k++)
			{
				printf("%d ", scval[j][k]);
			}
			printf("\nscval[%d]=",j+1);
			for(k=0;k<rowwidth[j+1];k++)
			{
				printf("%d ", scval[j+1][k]);
			}
			*/
			temp=scval[j];	
			scval[j]=scval[j+1];
			scval[j+1]=temp;
			temp=sccol[j];
			sccol[j]=sccol[j+1];
			sccol[j+1]=temp;
			tint=rowwidth[j];
			rowwidth[j]=rowwidth[j+1];
			rowwidth[j+1]=tint;
			tint=vecX[j];
			vecX[j]=vecX[j+1];
			vecX[j+1]=tint;
		}
	}
	}	
	
	for(i=0;i<N;i++)
	{
		if(scval[i][0]==0)
		{
			break;	
		}	
	}
	N=i;
		
	printf("\nmaxrowwidth=%d\n",maxrowwidth);
	printmat(scval,N,N);
//	printtofile(scval,N,"scval.txt");
//	printtofile(sccol,N,"sccol.txt");
	printf("\n Vector is:\n");
	for (i=0;i<N;i++)
	{
		printf("%d\n",vecX[i]);
	}
	

        
        //printmatscreen<<<1,1>>>(dev_b,N);
	multiply<<<N,N>>>(dev_a, dev_b, dev_c, N, N);
	
	hipMemcpy(result, dev_c, sizeof(int)*N, hipMemcpyDeviceToHost);

        for (i=0;i<N;i++)
        {
        	printf("\n%d",result[i]);
        }
        
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
	
	// NEED TO FIGURE OUT A WAY TO POPULATE cols SO AS TO HAVE varmat CREATED PROPERLY. SYSTEM CRASHES OTHERWISE
	
	int c=2;
	int* cols=Make1DIntArray(N/c);
	j=0;
	int colsum=0;
	for(i=0;i<N;i=i+c)
	{
		cols[j]=rowwidth[i];
		colsum+=cols[j];
		j++;
	}
	
	int** varscval=Changeto2DVariableIntArray(scval,N,N/c,c,cols);
	int** varsccol=Changeto2DVariableIntArray(sccol,N,N/c,c,cols);
	for (i=0;i<N/c;i++)
	{
		for(j=0;j<c;j++)
		{
			printf("\n");
			for (k=0;k<cols[i];k++)
			{
				printf("%d ",varscval[i*c+j][k]);
				printf("%d ",varsccol[i*c+j][k]);
				
			}
		}
	}
	int varsize=colsum*c;
	
	hipMalloc((void**)&dev_vec, sizeof(int)*N);
        hipMalloc((void**)&dev_scval_flat, sizeof(int)*varsize);
        hipMalloc((void**)&dev_result, sizeof(int)*N);
        hipMalloc((void**)&dev_sccol_flat, sizeof(int)*varsize);	
		
	hipMemcpy(dev_a, vecX, sizeof(int)*N, hipMemcpyHostToDevice);
        hipMemcpy(dev_b, varscval, sizeof(int)*varsize, hipMemcpyHostToDevice);
        hipMemcpy(dev_c, result, sizeof(int)*N, hipMemcpyHostToDevice);
        hipMemcpy(dev_d, varsccol, sizeof(int)*varsize, hipMemcpyHostToDevice);
	
}


        
        


