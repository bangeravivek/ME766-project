#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<unistd.h>
#include<time.h>

__global__ void multiply(int *scval, int *sccol, int *vec, int *result, int *cols, int *rowptr)
{
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
        int sum=0;
        int i;
        printf("\n tid=%d", tid);
   	for(i=0;i<cols[tid];i++)
	{
		sum += vec[sccol[rowptr[tid]+i]]*scval[rowptr[tid]+i];
		printf("\nrowptr[%d]=%d",tid, rowptr[tid]);
		printf("\n%d*%d=%d",scval[rowptr[tid]+i],vec[sccol[rowptr[tid]+i]],vec[sccol[rowptr[tid]+i]]*scval[rowptr[tid]+i]);
		printf("\nsccol[%d]=%d",rowptr[tid]+i, sccol[rowptr[tid]+i]);
		printf("\nvec[%d]=%d",sccol[rowptr[tid]+i], vec[sccol[rowptr[tid]+i]]);
		printf("\nSum=%d", sum);
		printf("\n");
		
	}
     	__syncthreads();
   	result[tid]=sum;
}

__global__ void printmatscreen(int* mat, int N)
{
	int i,j;
	for (i=0;i<N;i++)
	{	
		printf("%d ",mat[i]);
		
	}
	printf("\n");
}


int** Make2DIntArray(int arraySizeX, int arraySizeY)
{
	int** theArray;
	theArray = (int**) malloc(arraySizeX*sizeof(int*));
	int i;
	for (i = 0; i < arraySizeX; i++)
	   theArray[i] = (int*) malloc(arraySizeY*sizeof(int));
	int j;
	for (i=0;i<arraySizeX;i++)
	{
	    for (j=0;j<arraySizeY;j++)
	    {
	        theArray[i][j]=0;
	    }
	}
	return theArray;
}

int** Make2DVariableIntArray(int rows, int blocks, int blocksize, int* columns)
{
int** theArray;
theArray = (int**) malloc(rows*sizeof(int*));
int i, j, k;
for (i = 0; i < blocks; i++)
{
	k=columns[i];
	for (j=0; j < blocksize; j++)
	{
		theArray[i*blocksize+j] = (int*) malloc(k*sizeof(int));
	}
   
}
//int j;

for (i=0;i<blocks;i++)
{
    for (j=0;j<blocksize;j++)
    {
    	for (k=0;k<columns[i];k++)
	{
	        theArray[i*blocksize+j][k]=0;
    	}
    }
}

   return theArray;
}

int** Changeto2DVariableIntArray(int** theArray,int rows, int blocks, int blocksize, int* columns)
{

int** NewArray=Make2DVariableIntArray(rows,blocks,blocksize,columns);
int i, j, k;

for (i=0;i<blocks;i++)
{
    for (j=0;j<blocksize;j++)
    {
        for (k=0;k<columns[i];k++)
        {
                NewArray[i*blocksize+j][k]=theArray[i*blocksize+j][k];
        }
    }
}
printf("changed to multiple matrixes");
   return NewArray;
}

void init_zeros(int** matrix, int N)
{
	int i,j;
	for (i=0;i<N;i++)
	{	
		for (j=0;j<N;j++)
		{
			matrix[i][j]=0;
		}
	}
}

void printmat(int** matrix, int N, int Nj)
{
	int i,j;
	
	for (i=0;i<N;i++)
	{	
		printf("\n");
		for (j=0;j<N;j++)
		{
			printf("%d ",matrix[i][j]);
		}
	}
	printf("\n");
}

void printtofile(int** matrix, int K, char* filename)
{
	/*
	Prints original 2D matrices to file
	*/
	FILE *fp;
	fp=fopen(filename,"wt");
	int i,j;
	for (i=0;i<K;i++)
	{
		fprintf(fp, "\n");
		for (j=0;j<K;j++)
		{
			fprintf(fp, "%d\t", matrix[i][j]);
		}
	}
}

void printtofile1D(int* matrix, int K, char* filename)
{
	/*
	Prints resultant matrix to a file
	*/
	FILE *fp;
	fp=fopen(filename,"wt");
	int i,j;
	int counters=0;
	for (i=0;i<K;i++)
	{
		fprintf(fp, "\n");
		for (j=0;j<K;j++)
		{
			fprintf(fp, "%d \t", matrix[counters]);
			counters++;
		}
	}
}

int* Make1DIntArray(int arraySizeX)
{
int* theArray;
theArray = (int*)malloc(arraySizeX*sizeof(int));
int i;
for (i=0;i<arraySizeX;i++)
{
    theArray[i]=0;
}

   return theArray;
}

void freese(int sizeX, int sizeY, double** ptr)
{
    int i;
     for (i=0;i<sizeX;i++)
        free(ptr[i]);
    free(ptr);
}

int main()
{
	int N=6;
	
//	const int Dsize=1000;
	FILE *arr, *vec;
	int i,j,maxrowwidth=0,tint=0;
	int** a=Make2DIntArray(N,N);
//	int* val=Make1DIntArray(Dsize);
//	int* col=Make1DIntArray(Dsize);
//	int* row=Make1DIntArray(Dsize);
	int* result=Make1DIntArray(N);
	int* vecX=Make1DIntArray(N);
	int** scval=Make2DIntArray(N,N);    //sell c value
	int** sccol=Make2DIntArray(N,N);	//sell c col
	int* rowwidth=Make1DIntArray(N);	//number of elements in each row
	int* temp=Make1DIntArray(N);
	int *dev_vec, *dev_scval, *dev_result, *dev_sccol, *dev_cols, *dev_rowptr;
	
	//int val[10],col[10],row[10];
	arr=fopen("mat.txt","r");
	int k=0,cinrow=0;
//	struct timeval start, end;
//	gettimeofday(&start, NULL);
	
	
	//Reading the vector
	
	vec=fopen("vec.txt","r");
	for (i=0;i<N;i++)
	{
		fscanf(vec,"%d",&vecX[i]);
	}
	printf("\n Vector is:\n");
	for (i=0;i<N;i++)
	{
		printf("%d\n",vecX[i]);
	}
	
	//Reading the matrix
	
	for(i=0;i<N;i++)
	{	
		printf("\n");
		for(j=0;j<N;j++)
		{
			fscanf(arr,"%d",&a[i][j]);
			printf("%d ",a[i][j]);
		}
	}
	printf("\n");
	//row[i]=k;
       	//printf("\n k = %d\n ", k);
       	//sleep(10);
	//gettimeofday(&end, NULL);

	//double delta = ((end.tv_sec  - start.tv_sec) * 1000000u + 
	 //        end.tv_usec - start.tv_usec) / 1.e6;

//	printf("\nTime spent=%f\n", delta);	

	
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{	
			if(a[i][j]!=0)
			{
				scval[i][k]=a[i][j];
				//printf("\n scval[%d][%d]=%d",i,k,scval[i][k]);

				//sleep(1);
				sccol[i][k]=j;
				//printf("\n sccol[%d][%d]=%d",i,k,sccol[i][k]);
				rowwidth[i]=k+1;
				if(rowwidth[i]>maxrowwidth)
				{	//printf("\nrow[%d] width=%d\n",i,maxrowwidth);
					maxrowwidth=rowwidth[i];
				}k++;
			}
				
		}
		//printf("\nRow width %d = %d", i, rowwidth[i]);
		k=0;
	}
	
	
	for(i=0;i<N-1;i++)
	{
	for(j=0;j<N-1;j++)
	{
		if(rowwidth[j]<rowwidth[j+1])
		{	/*printf("\nrow %d width=%d",j,rowwidth[j]);
			printf("\nscval[%d]=",j);
			for(k=0;k<rowwidth[j];k++)
			{
				printf("%d ", scval[j][k]);
			}
			printf("\nscval[%d]=",j+1);
			for(k=0;k<rowwidth[j+1];k++)
			{
				printf("%d ", scval[j+1][k]);
			}
			*/
			temp=scval[j];	
			scval[j]=scval[j+1];
			scval[j+1]=temp;
			temp=sccol[j];
			sccol[j]=sccol[j+1];
			sccol[j+1]=temp;
			tint=rowwidth[j];
			rowwidth[j]=rowwidth[j+1];
			rowwidth[j+1]=tint;
			
		}
	}
	}	
	
	for(i=0;i<N;i++)
	{
		if(scval[i][0]==0)
		{
			break;	
		}	
	}
	if(i%2==1)
		N=i+1;
	else
		N=i;	
	printf("\nmaxrowwidth=%d\n",maxrowwidth);
	printmat(scval,N,N);
	printtofile(scval,N,"scval.txt");
	printtofile(sccol,N,"sccol.txt");
	printf("\n Vector is:\n");
	for (i=0;i<N;i++)
	{
		printf("%d\n",vecX[i]);
	}
	

        
        //printmatscreen<<<1,1>>>(dev_b,N);
	
	
	

        
       
	// NEED TO FIGURE OUT A WAY TO POPULATE cols SO AS TO HAVE varmat CREATED PROPERLY. SYSTEM CRASHES OTHERWISE
	
	int c=2;
	int* cols=Make1DIntArray(N/c);
	j=0;
	int colsum=0;
	for(i=0;i<N;i=i+c)
	{
		cols[j]=rowwidth[i];
		printf("\n cols[%d]=%d",j,cols[j]);
		colsum+=cols[j];
		j++;
	}
	
	int** varscval=Changeto2DVariableIntArray(scval,N,N/c,c,cols);
	int** varsccol=Changeto2DVariableIntArray(sccol,N,N/c,c,cols);
	
	for (i=0;i<N/c;i++)
	{
		for(j=0;j<c;j++)
		{
			printf("\n");
			for (k=0;k<cols[i];k++)
			{
				printf("%d ",varscval[i*c+j][k]);
				printf("%d \t",varsccol[i*c+j][k]);
				
			}
		}
	}
	int varsize=colsum*c;
	
	//flattening scval and sccol
	int counters=0;
	int* scval_flat=Make1DIntArray(varsize);
	int* sccol_flat=Make1DIntArray(varsize);
	int* rowptr=Make1DIntArray(N+1);
	rowptr[0]=0;	
	int countcols=0;
	int z=0;
	for (i=0;i<N/c;i++)
	{
		for(j=0;j<c;j++)
		{
			printf("\n");
			countcols=0;
			for (k=0;k<cols[i];k++)
			{
				
				scval_flat[counters]=varscval[i*c+j][k];
				if (scval_flat[counters]!=0)
				{
					sccol_flat[counters]=varsccol[i*c+j][k];
				}
				counters=counters+1;
				countcols=countcols+1;
			}
			rowptr[z+1]=rowptr[z]+countcols;
			z=z+1;
		}
	}
	printf("\n rowptrs:\n");
	for(i=0;i<N;i++)
		printf("%d ",rowptr[i]);
	printf("\n");
	
	hipMalloc((void**)&dev_vec, sizeof(int)*N);
        hipMalloc((void**)&dev_scval, sizeof(int)*varsize);
        hipMalloc((void**)&dev_result, sizeof(int)*N);
        hipMalloc((void**)&dev_sccol, sizeof(int)*varsize);	
        hipMalloc((void**)&dev_cols, sizeof(int)*(N/c));
        hipMalloc((void**)&dev_rowptr, sizeof(int)*N);
		
	hipMemcpy(dev_vec, vecX, sizeof(int)*N, hipMemcpyHostToDevice);
        hipMemcpy(dev_scval, scval_flat, sizeof(int)*varsize, hipMemcpyHostToDevice);
        hipMemcpy(dev_result, result, sizeof(int)*N, hipMemcpyHostToDevice);
        hipMemcpy(dev_sccol, sccol_flat, sizeof(int)*varsize, hipMemcpyHostToDevice);
        hipMemcpy(dev_cols, rowwidth, sizeof(int)*N, hipMemcpyHostToDevice);
        hipMemcpy(dev_rowptr, rowptr, sizeof(int)*N, hipMemcpyHostToDevice);
	
	printmatscreen<<<1,1>>>(dev_scval,varsize);
	printmatscreen<<<1,1>>>(dev_sccol,varsize);
	
	multiply<<<N/c,c>>>(dev_scval, dev_sccol, dev_vec, dev_result, dev_cols, dev_rowptr);
	
	hipMemcpy(result, dev_result, sizeof(int)*N, hipMemcpyDeviceToHost);
        for (i=0;i<N;i++)
        {
        	printf("\n%d",result[i]);
        }	
	hipFree(dev_vec);
        hipFree(dev_scval);
        hipFree(dev_result);
	hipFree(dev_sccol);
	hipFree(dev_cols);
	return 0;
	
}


        
        


